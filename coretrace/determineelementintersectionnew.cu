#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "types.h"
#include "gpu_proc.cuh"

__device__
void DetermineElementIntersectionNew(
            ElementInfo *Element,
            double PosRayIn[3],
            double CosRayIn[3],
            double PosRayOut[3],
            double CosRayOut[3],
            double DFXYZ[3],
            double *PathLength,
            int *ErrorFlag,
            int *Intercept,
            int *BacksideFlag )
{
    double r = 0.0, Ro = 0.0, Ri = 0.0, XL = 0.0, x = 0.0, y = 0.0;
    double /*SLOP30 = 0.0, SLOP60 = 0.0,*/ Y1 = 0.0, Y2 = 0.0, Y3 = 0.0, Y4 = 0.0;
    double P1x = 0.0, P1y = 0.0, P2x = 0.0, P2y = 0.0, P3x = 0.0, P3y = 0.0, P4x = 0.0, P4y = 0.0;
    //double Tn;
    int in_quad = 0;
   //ZAperPlane: real;

    *ErrorFlag = 0;
    double SLOP60 = 1.7320508075688767; //tan(60.0*(acos(-1.0)/180.0));

    //AperturePlane(Element);           <------- calculated now in ODConcentrator
    //ZAperPlane = Element->ZAperture;

    //find intersection with surface first
    Intersect(PosRayIn, CosRayIn, Element, PosRayOut, CosRayOut, DFXYZ, PathLength, ErrorFlag);
    if (*ErrorFlag > 0 || *PathLength < 0)
    {
        *Intercept = false;
        PosRayOut[0] = 0.0;
        PosRayOut[1] = 0.0;
        PosRayOut[2] = 0.0;
        CosRayOut[0] = 0.0;
        CosRayOut[1] = 0.0;
        CosRayOut[2] = 0.0;
        DFXYZ[0] = 0.0;
        DFXYZ[1] = 0.0;
        DFXYZ[2] = 0.0;
        *BacksideFlag = false;
        *PathLength = 0.0;
        goto Label_100;
    }

    x = PosRayOut[0];
    y = PosRayOut[1];
    r = sqrt(x*x + y*y);



    switch (Element->ShapeIndex)
    {
    case 'c':
    case 'C': //circular aperture
            Ro = Element->ParameterA/2.0;

            if (r > Ro) //ray falls outsideside circular aperture
            {
               *Intercept = false;
               PosRayOut[0] = 0.0;
               PosRayOut[1] = 0.0;
               PosRayOut[2] = 0.0;
               CosRayOut[0] = 0.0;
               CosRayOut[1] = 0.0;
               CosRayOut[2] = 0.0;
               DFXYZ[0] = 0.0;
               DFXYZ[1] = 0.0;
               DFXYZ[2] = 0.0;
               *PathLength = 0.0;
               *ErrorFlag = 0;
               *BacksideFlag = false;
               goto Label_100;
            }
            else
            {
                if (DOT_GPU(CosRayIn, DFXYZ) < 0)
                    *BacksideFlag = false;
                else
                    *BacksideFlag = true;
                *Intercept = true;
                goto Label_100;
            }

    case 'h':
    case 'H': //hexagonal aperture
            Ro = Element->ParameterA/2.0;

            if (r > Ro) //ray falls outside circular circumference aperture
            {
               *Intercept = false;
               PosRayOut[0] = 0.0;
               PosRayOut[1] = 0.0;
               PosRayOut[2] = 0.0;
               CosRayOut[0] = 0.0;
               CosRayOut[1] = 0.0;
               CosRayOut[2] = 0.0;
               DFXYZ[0] = 0.0;
               DFXYZ[1] = 0.0;
               DFXYZ[2] = 0.0;
               *PathLength = 0.0;
               *ErrorFlag = 0;
               *BacksideFlag = false;
               goto Label_100;
            }
            Ri = Ro*cos(30.0*(ACOSM1O180));

            if ( r <= Ri ) //ray falls inside inscribed circle
            {
                if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                    *BacksideFlag = false;
                else
                    *BacksideFlag = true;
                *Intercept = true;
                goto Label_100;
            }

            XL = sqrt(Ro*Ro - Ri*Ri); //otherwise break hexagon into 3 sections
            if ( (x <= Ro) && (x > XL) )  //1st section
            {
                Y1 = SLOP60*(x-Ro);
                Y2 = -Y1;
                if ( (y >= Y1) && (y <= Y2) )
                {
                    if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                        *BacksideFlag = false;
                    else
                        *BacksideFlag = true;

                    *Intercept = true;
                    goto Label_100;
                }
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

            if ( (x <= XL) && (x >= -XL) )    //2nd section
            {
                if ( (y >= -Ri) && (y <= Ri) )
                {
                    if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                        *BacksideFlag = false;
                    else
                        *BacksideFlag = true;
                    *Intercept = true;
                    goto Label_100;
                }
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

            if ( (x < -XL) && (x >= -Ro) )    //3rd section
            {
                Y3 = SLOP60*(x+Ro);
                Y4 = -Y3;
                if ( (y >= Y4) && (y <= Y3) )
                {
                    if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                        *BacksideFlag = false;
                    else
                        *BacksideFlag = true;
                    *Intercept = true;
                    goto Label_100;
                }
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

    case 't':
    case 'T': //Triangular aperture
            Ro = Element->ParameterA/2.0;

            if ( r > Ro ) //ray falls outside circular circumference aperture
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }
            Ri = Ro*sin(30.0*(ACOSM1O180));

            if ( r <= Ri )  //ray falls inside inscribed circle
            {
                if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                    *BacksideFlag = false;
                else
                    *BacksideFlag = true;
                *Intercept = true;
                goto Label_100;
            }

            if ( (x <= Ro) && (x > 0.0) )  //1st section
            {
                Y1 = -SLOP60*(x-Ri/cos(30.0*(ACOSM1O180)));
                Y2 = -Ri;
                if ( (y <= Y1) && (y >= Y2) )
                {
                    if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                        *BacksideFlag = false;
                    else
                        *BacksideFlag = true;
                    *Intercept = true;
                    goto Label_100;
                }
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }
            if ( (x >= -Ro) && (x <= 0.0) )  //2nd section
            {
                Y3 = SLOP60*(x+Ri/cos(30.0*(ACOSM1O180)));
                Y4 = -Ri;
                if ( (y >= Y4) && (y <= Y3) )
                {
                    if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                        *BacksideFlag = false;
                    else
                        *BacksideFlag = true;
                    *Intercept = true;
                    goto Label_100;
                }
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

    case 'r':
    case 'R': //Rectangular aperture

            if ( (x > Element->ParameterA/2.0) || (x < -Element->ParameterA/2.0) )
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

            if ( (y > Element->ParameterB/2.0) || (y < -Element->ParameterB/2.0) )
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

            if ( DOT_GPU(CosRayIn, DFXYZ) < 0 ) //successfully falls on rectangle
                *BacksideFlag = false;
            else
                *BacksideFlag = true;

            *Intercept = true;
            goto Label_100;


    case 'a':
    case 'A': //Annulus or torus contour

            if ( (Element->ParameterA == 0.0) && (Element->ParameterB == 0.0) ) goto Label_5; //torus

            if ( (r < Element->ParameterA) || (r > Element->ParameterB) )
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }
Label_5:
            if ( x >= 0.0 )
            {
                if ( (asin(y/r) > Element->ParameterC*(ACOSM1O180)/2.0) || (asin(y/r) < -Element->ParameterC*(ACOSM1O180)/2.0) )
                {
                    *Intercept = false;
                    PosRayOut[0] = 0.0;
                    PosRayOut[1] = 0.0;
                    PosRayOut[2] = 0.0;
                    CosRayOut[0] = 0.0;
                    CosRayOut[1] = 0.0;
                    CosRayOut[2] = 0.0;
                    DFXYZ[0] = 0.0;
                    DFXYZ[1] = 0.0;
                    DFXYZ[2] = 0.0;
                    *PathLength = 0.0;
                    *BacksideFlag = false;
                    *ErrorFlag = 0;
                    goto Label_100;
                }

                if ( DOT_GPU(CosRayIn, DFXYZ) < 0 ) //successfully falls on annular section
                    *BacksideFlag = false;
                else
                    *BacksideFlag = true;
                *Intercept = true;
                goto Label_100;
            }

            if ( x < 0.0 )
            {
                if ( (y >= 0) && ((acos(y/r)+M_PI/2.0) > Element->ParameterC*(ACOSM1O180)/2.0) )
                {
                    *Intercept = false;
                    PosRayOut[0] = 0.0;
                    PosRayOut[1] = 0.0;
                    PosRayOut[2] = 0.0;
                    CosRayOut[0] = 0.0;
                    CosRayOut[1] = 0.0;
                    CosRayOut[2] = 0.0;
                    DFXYZ[0] = 0.0;
                    DFXYZ[1] = 0.0;
                    DFXYZ[2] = 0.0;
                    *PathLength = 0.0;
                    *BacksideFlag = false;
                    *ErrorFlag = 0;
                    goto Label_100;
                }
                else if ( (y < 0) && ((-acos(-y/r)-M_PI/2.0) < -Element->ParameterC*(ACOSM1O180)/2.0) )
                {
                    *Intercept = false;
                    PosRayOut[0] = 0.0;
                    PosRayOut[1] = 0.0;
                    PosRayOut[2] = 0.0;
                    CosRayOut[0] = 0.0;
                    CosRayOut[1] = 0.0;
                    CosRayOut[2] = 0.0;
                    DFXYZ[0] = 0.0;
                    DFXYZ[1] = 0.0;
                    DFXYZ[2] = 0.0;
                    *PathLength = 0.0;
                    *BacksideFlag = false;
                    *ErrorFlag = 0;
                    goto Label_100;
                }

                if ( DOT_GPU(CosRayIn, DFXYZ) < 0 ) //successfully falls on annular section
                    *BacksideFlag = false;
                else
                    *BacksideFlag = true;
                *Intercept = true;
                goto Label_100;
            }

    case 'l':
    case 'L': //off axis aperture section of line focus trough  or cylinder
            if ( (Element->ParameterA == 0.0) && (Element->ParameterB == 0.0) ) goto Label_10; //for cylinder, only need to check for limits on y

            if ( (x < Element->ParameterA) || (x > Element->ParameterB) )
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

Label_10:
            if ( (y < -Element->ParameterC/2.0) || (y > Element->ParameterC/2.0) )
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *BacksideFlag = false;
                *ErrorFlag = 0;
                goto Label_100;
            }

            if ( DOT_GPU(CosRayIn, DFXYZ) < 0 ) //successfully falls on line focus or cylindrical section
                *BacksideFlag = false;
            else
                *BacksideFlag = true;

            *Intercept = true;
            goto Label_100;

    case 'i':
    case 'I': //irregular triangle
            P1x = Element->ParameterA;
            P1y = Element->ParameterB;
            P2x = Element->ParameterC;
            P2y = Element->ParameterD;
            P3x = Element->ParameterE;
            P3y = Element->ParameterF;

            if (!intri( P1x, P1y, P2x, P2y, P3x, P3y, x, y ))
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *ErrorFlag = 0;
                *BacksideFlag = false;
                goto Label_100;
            }

            if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                *BacksideFlag = false;
            else
                *BacksideFlag = true;
            *Intercept = true;
            goto Label_100;

    case 'q':
    case 'Q': //irregular quadrilateral
            P1x = Element->ParameterA;
            P1y = Element->ParameterB;
            P2x = Element->ParameterC;
            P2y = Element->ParameterD;
            P3x = Element->ParameterE;
            P3y = Element->ParameterF;
            P4x = Element->ParameterG;
            P4y = Element->ParameterH;

            in_quad = inquad(P1x, P1y, P2x, P2y, P3x, P3y, P4x, P4y, x, y);

            if (!in_quad)
            {
                *Intercept = false;
                PosRayOut[0] = 0.0;
                PosRayOut[1] = 0.0;
                PosRayOut[2] = 0.0;
                CosRayOut[0] = 0.0;
                CosRayOut[1] = 0.0;
                CosRayOut[2] = 0.0;
                DFXYZ[0] = 0.0;
                DFXYZ[1] = 0.0;
                DFXYZ[2] = 0.0;
                *PathLength = 0.0;
                *ErrorFlag = 0;
                *BacksideFlag = false;
                goto Label_100;
            }

            if ( DOT_GPU(CosRayIn, DFXYZ) < 0 )
                *BacksideFlag = false;
            else
                *BacksideFlag = true;
            *Intercept = true;
            goto Label_100;
    } //end select case

Label_100:
    if ( *BacksideFlag )   //if hit on backside of element then slope of surface is reversed
    {
        DFXYZ[0] = -DFXYZ[0];
        DFXYZ[1] = -DFXYZ[1];
        DFXYZ[2] = -DFXYZ[2];
    }
}
//End of Procedure--------------------------------------------------------------
